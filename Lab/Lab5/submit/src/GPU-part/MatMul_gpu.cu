#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"


int N = (1 << 8);
#define BLOCK_SIZE 16  // a matrix block size default 16×16
float bound = 100.0;
const float accuracy = 0.5;

__global__ void gemm_baseline(float *A, float *B, float *C, int N) {
    // one thread calcultate one element of C
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= N || j >= N) {
        return;
    }

    // suppose C[i][j] initializes with 0
    float tempc = 0.0f;  // use a local var to avoid many global memory access
    for (int k = 0; k < N; k++) {
        // C[i * N + j] += A[i * N + k] * B[k * N + j];
        tempc += A[i * N + k] * B[k * N + j];
    }
    C[i * N + j] = tempc;
}

void gemm_baseline_cpu(float *A, float *B, float *C) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            for (int k = 0; k < N; k++) {
                C[i * N + j] += A[i * N + k] * B[k * N + j];  // C[i][j] += A[i][k] * B[k][j];
            }
        }
    }
}

void gemm_verify(float *A, float *B, float *C) {
    /**
     * @brief verify avx correctness with baseline method
     * @param C has been calculated with avx-block method
     * now use baseline method calculate again and compare
     */
    float *C_baseline = (float *)malloc(N * N * sizeof(float));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            C_baseline[i * N + j] = 0.0;
        }
    }
    gemm_baseline_cpu(A, B, C_baseline);

    FILE *f = fopen("../../output/GPU-part/C_gpu_verify.txt", "w");
    if (f == NULL) {
        printf("cannot open output file\n");
    }
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fprintf(f, "%-20f ", C_baseline[i * N + j]);
        }
        fprintf(f, "\n");
    }
    fclose(f);

    // compare C and C_baseline
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (fabs(C[i * N + j] - C_baseline[i * N + j]) > accuracy) {
                printf("Wrong calculation in gpu-case!\n");
                return;
            }
        }
    }

    printf("GPU-case correctness verified.\n");
    free(C_baseline);
}

int main(int argc, char *argv[]) {
    if (argc == 2) {
        // with matrix size argument
        N = atoi(argv[1]);
    } else if (argc == 3) {
        // with matrix size and bound
        N = atoi(argv[1]);
        bound = atof(argv[2]);
    }

    // CPU malloc and initialization
    float *A = (float *)malloc(N * N * sizeof(float));
    float *B = (float *)malloc(N * N * sizeof(float));
    float *C = (float *)malloc(N * N * sizeof(float));
    // initialize A and B
    srand((unsigned int)time(NULL));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            double a = (-bound) + (1.0 * rand() / (1.0 * RAND_MAX)) * (2 * bound);
            double b = (-bound) + (1.0 * rand() / (1.0 * RAND_MAX)) * (2 * bound);
            A[i * N + j] = a;
            B[i * N + j] = b;
            C[i * N + j] = 0.0;
        }
    }

    // GPU malloc and initialization
    float *Ag, *Bg, *Cg;
    hipMalloc((void **)&Ag, sizeof(float) * N * N);
    hipMalloc((void **)&Bg, sizeof(float) * N * N);
    hipMalloc((void **)&Cg, sizeof(float) * N * N);
    // set gpu data using cpu initialization values
    hipMemcpy(Ag, A, sizeof(float) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(Bg, B, sizeof(float) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(Cg, C, sizeof(float) * N * N, hipMemcpyHostToDevice);

    // define grid size and block size
    int bN = BLOCK_SIZE;  // a thread block is bN×bN
    dim3 thread_per_block(bN, bN);
    dim3 block_per_grid(N / bN, N / bN);

    hipEvent_t start, end;
    float running_time;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    gemm_baseline<<<block_per_grid, thread_per_block>>>(Ag, Bg, Cg, N);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&running_time, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);
    printf("gpu-case running time: %lf ms\n", running_time);

    // copy the result from gpu to cpu
    hipMemcpy(C, Cg, sizeof(float) * N * N, hipMemcpyDeviceToHost);

    gemm_verify(A, B, C);

    FILE *f = fopen("../../output/GPU-part/C_gpu.txt", "w");
    if (f == NULL) {
        printf("cannot open output file\n");
        return 0;
    }
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fprintf(f, "%-20f ", C[i * N + j]);
        }
        fprintf(f, "\n");
    }
    fclose(f);

    // free memory
    free(A);
    free(B);
    free(C);
    hipFree(Ag);
    hipFree(Bg);
    hipFree(Cg);
    hipDeviceSynchronize();
    hipProfilerStop();
    return 0;
}