#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>


// 本机上每个线程块的共享内存大小为48KB
// 一个float为4B
// 共享内存最多可容纳12K个float
// 有三个矩阵, 假设留空一个矩阵的空间, 平均每个矩阵最多可用3KB
// 32×32 = 1024
// 线程块大小与矩阵块大小保持一致


int N = (1 << 8);
#define BLOCK_SIZE 8 // a matrix block size default 16×16
float bound = 100.0;
const float accuracy = 0.5;

__global__ void gemm_baseline(float *A, float *B, float *C, int N) {
    // one thread calcultate one element of C
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= N || j >= N) {
        return;
    }

    // suppose C[i][j] initializes with 0
    float tempc = 0.0f; // use a local var to avoid many global memory access
    for (int k = 0; k < N; k++) {
        // C[i * N + j] += A[i * N + k] * B[k * N + j];
        tempc += A[i * N + k] * B[k * N + j];
    }
    C[i * N + j] = tempc;
}

__global__ void gemm_blocking(float *A, float *B, float *C, int N) {
    // one thread calculate one element of C, i.e c[i][j]

    // malloc shared memory
    __shared__ float sharedA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sharedB[BLOCK_SIZE][BLOCK_SIZE];

    int tx = threadIdx.x, ty = threadIdx.y; // thread index in the block
    int i = blockIdx.x * blockDim.x + threadIdx.x; // element index of C
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    // bi, bj, bk denotes block i, j, k
    // now calculating block C[bi][bj]
    int bi = blockIdx.x;
    int bj = blockIdx.y;


    if (i >= N || j >= N) {
        return;
    }

    float sum = 0.0; // c[i][j]
    for (int bk = 0; bk < N / BLOCK_SIZE; bk++) {
        // block C[bi][bj] = summation_bk{ A[bi][bk] * B[bk][bj] }
        // there are N/BLOCK_SIZE blocks in a row/column
        // bk denotes it is the bk-th block in the row of A anf the column of B
        
        // load the block A[bi][bk] and B[bk][bj]
        int Ai = bi * BLOCK_SIZE + tx; // element index of A
        int Aj = bk * BLOCK_SIZE + ty;
        int Bi = bk * BLOCK_SIZE + tx; // element index of B
        int Bj = bj * BLOCK_SIZE + ty;
        sharedA[tx][ty] = A[Ai * N + Aj];
        sharedB[tx][ty] = B[Bi * N + Bj];
        __syncthreads(); // wait until the whole block A[bi][bk], B[bk][bj] loaded to the shared memory
        for (int tk = 0; tk < BLOCK_SIZE; tk++) {
            sum += sharedA[tx][tk] * sharedB[tk][ty];
        }
        __syncthreads(); // wait until all threads finished then next bk
    }

    C[i * N + j] = sum;
}

void gemm_baseline_cpu(float *A, float *B, float *C) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            for (int k = 0; k < N; k++) {
                C[i * N + j] += A[i * N + k] * B[k * N + j]; // C[i][j] += A[i][k] * B[k][j];
            }
        }
    }
}

void gemm_verify(float *A, float *B, float *C) {
    /**
     * @brief verify avx correctness with baseline method
     * @param C has been calculated with avx-block method
     * now use baseline method calculate again and compare
     */
    float *C_baseline = (float *)malloc(N * N * sizeof(float));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            C_baseline[i * N + j] = 0.0;
        }
    }
    gemm_baseline_cpu(A, B, C_baseline);

    FILE *f = fopen("../../output/GPU-part/C_gpu_blocking_verify.txt", "w");
    if (f == NULL) {
        printf("cannot open output file\n");
    }
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fprintf(f, "%-20f ", C_baseline[i * N + j]);
        }
        fprintf(f, "\n");
    }
    fclose(f);

    // compare C and C_baseline
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (fabs(C[i * N + j] - C_baseline[i * N + j]) > accuracy) {
                printf("Wrong calculation in gpu-blocking-case!\n");
                return;
            }
        }
    }

    printf("GPU-blocking-case correctness verified.\n");
    free(C_baseline);
}

int main(int argc, char *argv[]) {
    if (argc == 2) {
        // with matrix size argument
        N = atoi(argv[1]);
    } else if (argc == 3) {
        // with matrix size and bound
        N = atoi(argv[1]);
        bound = atof(argv[2]);
    }

    // CPU malloc and initialization
    float *A = (float *)malloc(N * N * sizeof(float));
    float *B = (float *)malloc(N * N * sizeof(float));
    float *C = (float *)malloc(N * N * sizeof(float));
    // initialize A and B
    srand((unsigned int)time(NULL));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            double a = (-bound) + (1.0 * rand() / (1.0 * RAND_MAX)) * (2 * bound);
            double b = (-bound) + (1.0 * rand() / (1.0 * RAND_MAX)) * (2 * bound);
            A[i * N + j] = a;
            B[i * N + j] = b;
            C[i * N + j] = 0.0;
        }
    }


    // GPU malloc and initialization
    float *Ag, *Bg, *Cg;
    hipMalloc((void**)&Ag, sizeof(float) * N * N);
    hipMalloc((void**)&Bg, sizeof(float) * N * N);
    hipMalloc((void**)&Cg, sizeof(float) * N * N);
    // set gpu data using cpu initialization values
    hipMemcpy(Ag, A, sizeof(float) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(Bg, B, sizeof(float) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(Cg, C, sizeof(float) * N * N, hipMemcpyHostToDevice);


    // define grid size and block size
    int bN = BLOCK_SIZE; // a thread block is bN×bN
    dim3 thread_per_block(bN, bN);
    dim3 block_per_grid(N/bN, N/bN);

    hipEvent_t start, end;
    float running_time;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    gemm_blocking<<<block_per_grid, thread_per_block>>>(Ag, Bg, Cg, N);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&running_time, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);
    printf("gpu-blocking-case running time: %lf ms\n", running_time);

    // copy the result from gpu to cpu
    hipMemcpy(C, Cg, sizeof(float) * N * N, hipMemcpyDeviceToHost);

    gemm_verify(A, B, C);




    // to compare baseline and blocking
    hipEvent_t start1, end1;
    float running_time1;
    hipEventCreate(&start1);
    hipEventCreate(&end1);
    hipEventRecord(start1, 0);

    gemm_baseline<<<block_per_grid, thread_per_block>>>(Ag, Bg, Cg, N);

    hipEventRecord(end1, 0);
    hipEventSynchronize(end1);
    hipEventElapsedTime(&running_time1, start1, end1);
    hipEventDestroy(start1);
    hipEventDestroy(end1);
    printf("gpu-baseline running time: %lf ms\n", running_time1);



    FILE *f = fopen("../../output/GPU-part/C_gpu_blocking.txt", "w");
    if (f == NULL) {
        printf("cannot open output file\n");
        return 0;
    }
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fprintf(f, "%-20f ", C[i * N + j]);
        }
        fprintf(f, "\n");
    }
    fclose(f);

    // free memory
    free(A);
    free(B);
    free(C);
    hipFree(Ag);
    hipFree(Bg);
    hipFree(Cg);
    hipDeviceSynchronize();
    hipProfilerStop();
    return 0;
}